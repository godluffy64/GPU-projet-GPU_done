#include "hip/hip_runtime.h"
#include "utils/commonCUDA.hpp"
#include "utils/chronoGPU.hpp"

#include "carteGPU.hpp"
#include <cmath>
#include <iostream>
#include <cstdlib>
#include <iomanip>

using namespace std;

/*__global__ void kernelMap(uint8_t *h_in, uint8_t *h_out, const int MapWidth, const int MapHeight, const int Cx, const int Cy)
{
    int Px = blockIdx.x * blockDim.x + threadIdx.x;
    int Py = blockIdx.y * blockDim.y + threadIdx.y;

    if (Px < MapWidth && Py < MapHeight)
    {
        float Dx = Px - Cx;
        float Dy = Py - Cy;
        float Dz = h_in[Py * MapWidth + Px] - h_in[Cy * MapWidth + Cx];
        float D = fmaxf(fabsf(Dx), fabsf(Dy));
        double angle_ref = atan2f(Dz, sqrt((Dx * Dx) + (Dy * Dy)));

        float Cx_dda = static_cast<float>(Cx), Cy_dda = static_cast<float>(Cy);
        float incX = Dx / D;
        float incY = Dy / D;

        h_out[Py * MapWidth + Px] = 244;

        for (int i = 0; i < D - 1; i++)
        {
            Cx_dda += incX;
            Cy_dda += incY;
            int Lx = static_cast<int>(round(Cx_dda));
            int Ly = static_cast<int>(round(Cy_dda));

            Dx = Px - Lx;
            Dy = Py - Ly;
            Dz = h_in[Py * MapWidth + Px] - h_in[Ly * MapWidth + Lx];

            double angle = atan2f(Dz, sqrt((Dx * Dx) + (Dy * Dy)));

            if (angle_ref >= angle)
            {
                h_out[Py * MapWidth + Px] = 0;
                break;
            }
        }
    }
}*/

__global__ void kernelMap(uint8_t *h_in, uint8_t *h_out, const int MapWidth, const int MapHeight, const int Cx, const int Cy)
{

    for(int Py = blockDim.y * blockIdx.y + threadIdx.y; Py < MapHeight; Py += blockDim.y * gridDim.y)
    {
        for(int Px = blockDim.x * blockIdx.x + threadIdx.x; Px < MapWidth; Px += blockDim.x * gridDim.x)
        {
        // DDA entre le point c (Cx, Cy) et le point P (indexX, indexY);
            
            float Dx = Px - Cx;
            float Dy = Py - Cy;
            float Dz = h_in[Py * MapWidth + Px] - h_in[Cy * MapWidth + Cx];
            float D = max(abs(Dx), abs(Dy));
            double angle_ref = atan(Dz / sqrt((Dx * Dx) + (Dy * Dy)));

            float Cx_dda = static_cast<float>(Cx), Cy_dda = static_cast<float>(Cy);
            float incX = Dx / D;
            float incY = Dy / D;

            h_out[Py * MapWidth + Px] = 244;

            for (int i = 0; i < D - 1; i++)
            {
                Cx_dda += incX;
                Cy_dda += incY;
                int Lx = static_cast<int>(round(Cx_dda));
                int Ly = static_cast<int>(round(Cy_dda));

                Dx = Px - Lx;
                Dy = Py - Ly;
                Dz = h_in[Py * MapWidth + Px] - h_in[Ly * MapWidth + Lx];

                double angle = atan(Dz / sqrt((Dx * Dx) + (Dy * Dy)));

                if (angle_ref >= angle)
                {
                    h_out[Py * MapWidth + Px] = 0;
                    break;
                }                     
            } 
        }
    }
}

void carteGPU(uint8_t *h_in, uint8_t *h_out, const int MapWidth, const int MapHeight, const int Cx, const int Cy)
{
    uint8_t *d_in, *d_out;
    hipMalloc((void**)&d_in, MapWidth * MapHeight * sizeof(uint8_t));
    hipMalloc((void**)&d_out, MapWidth * MapHeight * sizeof(uint8_t));

    // Copie des données vers le GPU
    hipMemcpy(d_in, h_in, MapWidth * MapHeight * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Définir la taille des blocs et des grilles
    dim3 blockSize(16, 16);
    dim3 gridSize((MapWidth + blockSize.x - 1) / blockSize.x, (MapHeight + blockSize.y - 1) / blockSize.y);

    // Appel du kernel
    kernelMap<<<gridSize, blockSize>>>(d_in, d_out, MapWidth, MapHeight, Cx, Cy);

    // Copie des résultats depuis le GPU
    hipMemcpy(h_out, d_out, MapWidth * MapHeight * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Libération de la mémoire GPU
    hipFree(d_in);
    hipFree(d_out);
}